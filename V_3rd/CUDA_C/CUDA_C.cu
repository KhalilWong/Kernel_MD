#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <fstream>
#include <algorithm>
using namespace std;
//const int Lib_N = 100000;
//const int SampleParallel_N = 1000;
const int Atom_type_N = 2;
const int Pair_type_N = 3;
const int Pt_I = 6, Pt_J = 6, Pt_K = 3;
const int Pt_N = 4 * Pt_I * Pt_J * Pt_K;
const int Ar_N = 1;

class Parameters {
public:
	int Rt, Tt, dumpstep;
	double Mass[Atom_type_N], T[Atom_type_N], mp_V[Atom_type_N], LJ_E[Pair_type_N], LJ_S[Pair_type_N], Box_x[2], Box_y[2], Box_z[2], Pt_ePos_x[Pt_N], Pt_ePos_y[Pt_N], Pt_ePos_z[Pt_N], Pt_argVel[3];
	double PI, kB, fcc_lattice, nd_Mass, nd_Energy, nd_Length, nd_Velocity, nd_Time, nd_Acceleration, cutoff, d, spr_k, dt, Pt_T;
	bool state;
	void Init();
	void Initialization(int All_type[], double All_Pos_x[], double All_Pos_y[], double All_Pos_z[], double All_Vel_x[], double All_Vel_y[], double All_Vel_z[], double All_Acc_x[], double All_Acc_y[], double All_Acc_z[]);
	void Initialization_Kernel(int All_type[], double All_Pos_x[], double All_Pos_y[], double All_Pos_z[], double All_Vel_x[], double All_Vel_y[], double All_Vel_z[], double All_Acc_x[], double All_Acc_y[], double All_Acc_z[]);
	void rescale_T1(double All_Vel_x[], double All_Vel_y[], double All_Vel_z[]);
	void rescale_T3(double All_Vel_x[], double All_Vel_y[], double All_Vel_z[]);
	void Dump(int All_type[], double All_Pos_x[], double All_Pos_y[], double All_Pos_z[], int timestep, int ds = 1);
	void Exit(int All_type[], double All_Pos_x[], double All_Pos_y[], double All_Pos_z[], int timestep);
	double random();
};
void Parameters::Init() {
	//
	PI = 3.14159265;
	kB = 1.38E-23;
	Mass[0] = 39.95 / 6.02 * 1E-26;//kg
	Mass[1] = 195.08 / 6.02 * 1E-26;
	LJ_E[0] = 1.654E-21;//J
	LJ_E[1] = 5.207E-20;
	LJ_E[2] = 1.093E-21;
	LJ_S[0] = 3.40 * 1E-10;//m
	LJ_S[1] = 2.47 * 1E-10;
	LJ_S[2] = 2.94 * 1E-10;
	cutoff = 10 * 1E-10;
	fcc_lattice = 3.93E-10;
	T[0] = 300.;
	T[1] = 300.;
	mp_V[0] = sqrt(2 * kB*T[0] / Mass[0]);//
	mp_V[1] = sqrt(3 * kB*T[1] / Mass[1]);//
	//
	nd_Mass = Mass[1];
	nd_Energy = LJ_E[1];
	nd_Length = LJ_S[1];
	nd_Velocity = sqrt(nd_Energy / nd_Mass);
	nd_Time = nd_Length / nd_Velocity;
	nd_Acceleration = nd_Energy / (nd_Mass * nd_Length);
	//
	Mass[0] /= nd_Mass;
	Mass[1] /= nd_Mass;
	LJ_E[0] /= nd_Energy;
	LJ_E[1] /= nd_Energy;
	LJ_E[2] /= nd_Energy;
	LJ_S[0] /= nd_Length;
	LJ_S[1] /= nd_Length;
	LJ_S[2] /= nd_Length;
	cutoff /= nd_Length;
	fcc_lattice /= nd_Length;
	mp_V[0] /= nd_Velocity;
	mp_V[1] /= nd_Velocity;
	d = 5.0;
	spr_k = 5000.;
	dt = 0.001;
	Rt = 100;
	Tt = 35;
	dumpstep = 1;
	//
	Box_x[0] = 0;
	Box_x[1] = Pt_I * fcc_lattice;
	Box_y[0] = 0;
	Box_y[1] = Pt_J * fcc_lattice;
	Box_z[0] = -(Pt_K - 0.5)*fcc_lattice;
	Box_z[1] = d;
	//
	state = true;

	cout << "*******Parameters Initialized!*******\n";
}

/******************************************************************************/
void Parameters::Initialization(int All_type[], double All_Pos_x[], double All_Pos_y[], double All_Pos_z[], double All_Vel_x[], double All_Vel_y[], double All_Vel_z[], double All_Acc_x[], double All_Acc_y[], double All_Acc_z[]) {
	int i;
	double *d_Pt_argVel, *d_Pt_T, *d_Mass, *d_T, *d_LJ_E, *d_LJ_S, *d_cutoff, *d_spr_k;
	double *d_All_Pos_x, *d_All_Pos_y, *d_All_Pos_z, *d_All_Vel_x, *d_All_Vel_y, *d_All_Vel_z, *d_All_Acc_x, *d_All_Acc_y, *d_All_Acc_z, *d_Box_x, *d_Box_y, *d_Box_z, *d_Pt_ePos_x, *d_Pt_ePos_y, *d_Pt_ePos_z;
	__global__ void Pos_period(double *All_Pos_x, double *All_Pos_y, double *Box_x, double *Box_y, double *Pt_ePos_x, double *Pt_ePos_y);
	__global__ void rescale_T2(double *All_Vel_x, double *All_Vel_y, double *All_Vel_z, double *Pt_argVel);
	__global__ void rescale_T4(double *All_Vel_x, double *All_Vel_y, double *All_Vel_z, double *T, double *Pt_T);
	__global__ void Acceleration_period(double *All_Pos_x, double *All_Pos_y, double *All_Pos_z, double *All_Acc_x, double *All_Acc_y, double *All_Acc_z, double *LJ_E, double *LJ_S, double *Box_x, double *Box_y, double *Box_z, double *cutoff, double *Pt_ePos_x, double *Pt_ePos_y, double *Pt_ePos_z, double *spr_k, double *Mass);

	cout << "Box_ZoneX: " << Box_x[0] << ", " << Box_x[1] << "\n";
	cout << "Box_ZoneY: " << Box_y[0] << ", " << Box_y[1] << "\n";
	cout << "Box_ZoneZ: " << Box_z[0] << ", " << Box_z[1] << "\n";
	//
	Initialization_Kernel(All_type, All_Pos_x, All_Pos_y, All_Pos_z, All_Vel_x, All_Vel_y, All_Vel_z, All_Acc_x, All_Acc_y, All_Acc_z);
	//
	hipMalloc((void**)&d_All_Pos_x, sizeof(All_Pos_x));
	hipMalloc((void**)&d_All_Pos_y, sizeof(All_Pos_y));
	hipMalloc((void**)&d_All_Pos_z, sizeof(All_Pos_z));
	hipMalloc((void**)&d_All_Vel_x, sizeof(All_Vel_x));
	hipMalloc((void**)&d_All_Vel_y, sizeof(All_Vel_y));
	hipMalloc((void**)&d_All_Vel_z, sizeof(All_Vel_z));
	hipMalloc((void**)&d_All_Acc_x, sizeof(All_Acc_x));
	hipMalloc((void**)&d_All_Acc_y, sizeof(All_Acc_y));
	hipMalloc((void**)&d_All_Acc_z, sizeof(All_Acc_z));
	hipMalloc((void**)&d_Box_x, sizeof(Box_x));
	hipMalloc((void**)&d_Box_y, sizeof(Box_y));
	hipMalloc((void**)&d_Box_z, sizeof(Box_z));
	hipMalloc((void**)&d_Pt_ePos_x, sizeof(Pt_ePos_x));
	hipMalloc((void**)&d_Pt_ePos_y, sizeof(Pt_ePos_y));
	hipMalloc((void**)&d_Pt_ePos_z, sizeof(Pt_ePos_z));
	hipMalloc((void**)&d_Mass, sizeof(Mass));
	hipMalloc((void**)&d_T, sizeof(T));
	hipMalloc((void**)&d_LJ_E, sizeof(LJ_E));
	hipMalloc((void**)&d_LJ_S, sizeof(LJ_S));
	hipMalloc((void**)&d_cutoff, sizeof(double));
	hipMalloc((void**)&d_spr_k, sizeof(double));
	hipMalloc((void**)&d_Pt_argVel, sizeof(Pt_argVel));
	hipMalloc((void**)&d_Pt_T, sizeof(double));
	//
	hipMemcpy(d_All_Pos_x, All_Pos_x, sizeof(All_Pos_x), hipMemcpyHostToDevice);
	hipMemcpy(d_All_Pos_y, All_Pos_y, sizeof(All_Pos_y), hipMemcpyHostToDevice);
	hipMemcpy(d_All_Pos_z, All_Pos_z, sizeof(All_Pos_z), hipMemcpyHostToDevice);
	hipMemcpy(d_All_Vel_x, All_Vel_x, sizeof(All_Vel_x), hipMemcpyHostToDevice);
	hipMemcpy(d_All_Vel_y, All_Vel_y, sizeof(All_Vel_y), hipMemcpyHostToDevice);
	hipMemcpy(d_All_Vel_z, All_Vel_z, sizeof(All_Vel_z), hipMemcpyHostToDevice);
	hipMemcpy(d_All_Acc_x, All_Acc_x, sizeof(All_Acc_x), hipMemcpyHostToDevice);
	hipMemcpy(d_All_Acc_y, All_Acc_y, sizeof(All_Acc_y), hipMemcpyHostToDevice);
	hipMemcpy(d_All_Acc_z, All_Acc_z, sizeof(All_Acc_z), hipMemcpyHostToDevice);
	hipMemcpy(d_Box_x, Box_x, sizeof(Box_x), hipMemcpyHostToDevice);
	hipMemcpy(d_Box_y, Box_y, sizeof(Box_y), hipMemcpyHostToDevice);
	hipMemcpy(d_Box_z, Box_z, sizeof(Box_z), hipMemcpyHostToDevice);
	hipMemcpy(d_Pt_ePos_x, Pt_ePos_x, sizeof(Pt_ePos_x), hipMemcpyHostToDevice);
	hipMemcpy(d_Pt_ePos_y, Pt_ePos_y, sizeof(Pt_ePos_y), hipMemcpyHostToDevice);
	hipMemcpy(d_Pt_ePos_z, Pt_ePos_z, sizeof(Pt_ePos_z), hipMemcpyHostToDevice);
	hipMemcpy(d_Mass, Mass, sizeof(Mass), hipMemcpyHostToDevice);
	hipMemcpy(d_T, T, sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(d_LJ_E, LJ_E, sizeof(LJ_E), hipMemcpyHostToDevice);
	hipMemcpy(d_LJ_S, LJ_S, sizeof(LJ_S), hipMemcpyHostToDevice);
	hipMemcpy(d_cutoff, &cutoff, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_spr_k, &spr_k, sizeof(double), hipMemcpyHostToDevice);
	//
	Pos_period << <1, Pt_N + Ar_N >> >(d_All_Pos_x, d_All_Pos_y, d_Box_x, d_Box_y, d_Pt_ePos_x, d_Pt_ePos_y);
	hipDeviceSynchronize();
	//
	Box_z[0]=All_Pos_z[0];
	for(i=0;i<Pt_N;i++){
		if(All_Pos_z[i]<Box_z[0]){
			Box_z[0]=All_Pos_z[i];
		}
	}
	cout<<Box_z[0]<<"\n";
	hipMemcpy(d_Box_z, Box_z, sizeof(Box_z), hipMemcpyHostToDevice);
	//
	rescale_T1(All_Vel_x, All_Vel_y, All_Vel_z);
	hipMemcpy(d_Pt_argVel, Pt_argVel, sizeof(Pt_argVel), hipMemcpyHostToDevice);
	rescale_T2 << <1, Pt_N >> > (d_All_Vel_x, d_All_Vel_y, d_All_Vel_z, d_Pt_argVel);
	hipDeviceSynchronize();
	hipMemcpy(All_Vel_x, d_All_Vel_x, sizeof(All_Vel_x), hipMemcpyDeviceToHost);
	hipMemcpy(All_Vel_y, d_All_Vel_y, sizeof(All_Vel_y), hipMemcpyDeviceToHost);
	hipMemcpy(All_Vel_z, d_All_Vel_z, sizeof(All_Vel_z), hipMemcpyDeviceToHost);
	rescale_T3(All_Vel_x, All_Vel_y, All_Vel_z);
	hipMemcpy(d_Pt_T, &Pt_T, sizeof(double), hipMemcpyHostToDevice);
	rescale_T4 << <1, Pt_N >> > (d_All_Vel_x, d_All_Vel_y, d_All_Vel_z, d_T, d_Pt_T);
	hipDeviceSynchronize();
	//
	Acceleration_period << <1, Pt_N + Ar_N >> >(d_All_Pos_x, d_All_Pos_y, d_All_Pos_z, d_All_Acc_x, d_All_Acc_y, d_All_Acc_z, d_LJ_E, d_LJ_S, d_Box_x, d_Box_y, d_Box_z, d_cutoff, d_Pt_ePos_x, d_Pt_ePos_y, d_Pt_ePos_z, d_spr_k, d_Mass);
	hipDeviceSynchronize();
	hipMemcpy(All_Pos_x, d_All_Pos_x, sizeof(All_Pos_x), hipMemcpyDeviceToHost);
	hipMemcpy(All_Pos_y, d_All_Pos_y, sizeof(All_Pos_y), hipMemcpyDeviceToHost);
	hipMemcpy(All_Pos_z, d_All_Pos_z, sizeof(All_Pos_z), hipMemcpyDeviceToHost);
	hipMemcpy(All_Vel_x, d_All_Vel_x, sizeof(All_Vel_x), hipMemcpyDeviceToHost);
	hipMemcpy(All_Vel_y, d_All_Vel_y, sizeof(All_Vel_y), hipMemcpyDeviceToHost);
	hipMemcpy(All_Vel_z, d_All_Vel_z, sizeof(All_Vel_z), hipMemcpyDeviceToHost);
	hipMemcpy(All_Acc_x, d_All_Acc_x, sizeof(All_Acc_x), hipMemcpyDeviceToHost);
	hipMemcpy(All_Acc_y, d_All_Acc_y, sizeof(All_Acc_y), hipMemcpyDeviceToHost);
	hipMemcpy(All_Acc_z, d_All_Acc_z, sizeof(All_Acc_z), hipMemcpyDeviceToHost);
	hipMemcpy(Pt_ePos_x, d_Pt_ePos_x, sizeof(Pt_ePos_x), hipMemcpyDeviceToHost);
	hipMemcpy(Pt_ePos_y, d_Pt_ePos_y, sizeof(Pt_ePos_y), hipMemcpyDeviceToHost);
	hipMemcpy(Pt_ePos_z, d_Pt_ePos_z, sizeof(Pt_ePos_z), hipMemcpyDeviceToHost);
	//
	cout << "Created " << Pt_N << " Pt\n";
	cout << "Created " << Ar_N << " Ar\n";
	cout << "Pt Average Speed in X: " << Pt_argVelx << "\n";
	cout << "Pt Average Speed in Y: " << Pt_argVely << "\n";
	cout << "Pt Average Speed in Z: " << Pt_argVelz << "\n";
	cout << "Pt Temperature: " << Pt_T << "\n";
	cout << "Ar Incidence Speed: " << All_Vel_x[Pt_N] << "," << All_Vel_y[Pt_N] << "," << All_Vel_z[Pt_N] << "\n";
	cout << "*******Model Initialization Done!*******\n";
	hipFree(d_All_Pos_x);
	hipFree(d_All_Pos_y);
	hipFree(d_All_Pos_z);
	hipFree(d_All_Vel_x);
	hipFree(d_All_Vel_y);
	hipFree(d_All_Vel_z);
	hipFree(d_All_Acc_x);
	hipFree(d_All_Acc_y);
	hipFree(d_All_Acc_z);
	hipFree(d_Box_x);
	hipFree(d_Box_y);
	hipFree(d_Box_z);
	hipFree(d_Pt_ePos_x);
	hipFree(d_Pt_ePos_y);
	hipFree(d_Pt_ePos_z);
	hipFree(d_nd_Velocity);
	hipFree(d_Mass);
	hipFree(d_nd_Mass);
	hipFree(d_kB);
	hipFree(d_T);
	hipFree(d_LJ_E);
	hipFree(d_LJ_S);
	hipFree(d_cutoff);
	hipFree(d_spr_k);
	hipFree(d_Pt_argVel);
	hipFree(d_Pt_T);
}

/******************************************************************************/
void Parameters::Initialization_Kernel(int All_type[], double All_Pos_x[], double All_Pos_y[], double All_Pos_z[], double All_Vel_x[], double All_Vel_y[], double All_Vel_z[], double All_Acc_x[], double All_Acc_y[], double All_Acc_z[]) {
	int i, j, k, axis, count;
	double R1, R2, Rx, Ry;

	count = 0;
	srand((unsigned)time(NULL));
	for (i = 0; i < 2 * Pt_I; i++) {
		for (j = 0; j < 2 * Pt_J; j++) {
			for (k = 0; k < 2 * Pt_K; k++) {
				if (i / 2. + j / 2. + k / 2. == int(i / 2. + j / 2. + k / 2.)) {
					All_type[count] = 1;
					All_Pos_x[count] = i / 2.*fcc_lattice;
					Pt_ePos_x[count] = All_Pos_x[count];
					All_Pos_y[count] = j / 2.*fcc_lattice;
					Pt_ePos_y[count] = All_Pos_y[count];
					All_Pos_z[count] = (k / 2. - 2.5)*fcc_lattice;
					Pt_ePos_z[count] = All_Pos_z[count];
					R1 = random();
					R2 = random();
					All_Vel_x[count] = mp_V[1] / sqrt(3)*sqrt(-2 * log(R1))*cos(2 * PI*R2);
					R1 = random();
					R2 = random();
					All_Vel_y[count] = mp_V[1] / sqrt(3)*sqrt(-2 * log(R1))*cos(2 * PI*R2);
					R1 = random();
					R2 = random();
					All_Vel_z[count] = mp_V[1] / sqrt(3)*sqrt(-2 * log(R1))*cos(2 * PI*R2);
					All_Acc_x[count] = 0.0;
					All_Acc_y[count] = 0.0;
					All_Acc_z[count] = 0.0;
					count += 1;
				}
			}
		}
	}
	Rx = random();
	Ry = random();
	All_type[count] = 0;
	All_Pos_x[count] = Box_x[0] + (Box_x[1] - Box_x[0]) * Rx;
	All_Pos_y[count] = Box_y[0] + (Box_y[1] - Box_y[0]) * Ry;
	All_Pos_z[count] = Box_z[1];
	R1 = random();
	R2 = random();
	All_Vel_x[count] = mp_V[0] * sqrt(-log(R1))*cos(2 * PI*R2);//Maxwell�ֲ�
	R1 = random();
	R2 = random();
	All_Vel_y[count] = mp_V[0] * sqrt(-log(R1))*sin(2 * PI*R2);
	R1 = random();
	All_Vel_z[count] = -mp_V[0] * sqrt(-log(R1));
	All_Acc_x[count] = 0.0;
	All_Acc_y[count] = 0.0;
	All_Acc_z[count] = 0.0;
}

/******************************************************************************/
__global__ void Pos_period(double *All_Pos_x, double *All_Pos_y, double *Box_x, double *Box_y, double *Pt_ePos_x, double *Pt_ePos_y) {
	int tid = threadIdx.x;

	if (tid<Pt_N + Ar_N) {
		//X
		if (All_Pos_x[tid]<Box_x[0]) {
			All_Pos_x[tid] += Box_x[1] - Box_x[0];
			if (tid<Pt_N) {
				Pt_ePos_x[tid] += Box_x[1] - Box_x[0];
			}
		}
		else if (All_Pos_x[tid] >= Box_x[1]) {
			All_Pos_x[tid] -= Box_x[1] - Box_x[0];
			if (tid<Pt_N) {
				Pt_ePos_x[tid] -= Box_x[1] - Box_x[0];
			}
		}
		//Y
		if (All_Pos_y[tid]<Box_y[0]) {
			All_Pos_y[tid] += Box_y[1] - Box_y[0];
			if (tid<Pt_N) {
				Pt_ePos_y[tid] += Box_y[1] - Box_y[0];
			}
		}
		else if (All_Pos_y[tid] >= Box_y[1]) {
			All_Pos_y[tid] -= Box_y[1] - Box_y[0];
			if (tid<Pt_N) {
				Pt_ePos_y[tid] -= Box_y[1] - Box_y[0];
			}
		}
	}
}

/******************************************************************************/
void Parameters::rescale_T1(double All_Vel_x[], double All_Vel_y[], double All_Vel_z[]) {
	int i;

	Pt_argVel[0] = 0.0;
	Pt_argVel[1] = 0.0;
	Pt_argVel[2] = 0.0;
	for (i = 0; i < Pt_N; i++) {
		Pt_argVel[0] += All_Vel_x[i] / Pt_N;
		Pt_argVel[1] += All_Vel_y[i] / Pt_N;
		Pt_argVel[2] += All_Vel_z[i] / Pt_N;
	}
}

/******************************************************************************/
__global__ void rescale_T2(double *All_Vel_x, double *All_Vel_y, double *All_Vel_z, double *Pt_argVel) {
	int tid = threadIdx.x;

	if (tid<Pt_N) {
		All_Vel_x[tid] -= Pt_argVel[0];
		All_Vel_y[tid] -= Pt_argVel[1];
		All_Vel_z[tid] -= Pt_argVel[2];
	}
}

/******************************************************************************/
void Parameters::rescale_T3(double All_Vel_x[], double All_Vel_y[], double All_Vel_z[]) {
	int i;

	Pt_T = 0.0;
	for (i = 0; i < Pt_N; i++) {
		Pt_T += All_Vel_x[i] * All_Vel_x[i] + All_Vel_y[i] * All_Vel_y[i] + All_Vel_z[i] * All_Vel_z[i];
	}
	Pt_T *= nd_Velocity * nd_Velocity * Mass[1] * nd_Mass / (3 * Pt_N * kB);
}

/******************************************************************************/
__global__ void rescale_T4(double *All_Vel_x, double *All_Vel_y, double *All_Vel_z, double *T, double *Pt_T) {
	int tid = threadIdx.x;

	if (tid<Pt_N) {
		All_Vel_x[tid] *= sqrt(T[1] / (*Pt_T));
		All_Vel_y[tid] *= sqrt(T[1] / (*Pt_T));
		All_Vel_z[tid] *= sqrt(T[1] / (*Pt_T));
	}
}

/******************************************************************************/
__global__ void Acceleration_period(double *All_Pos_x, double *All_Pos_y, double *All_Pos_z, double *All_Acc_x, double *All_Acc_y, double *All_Acc_z, double *LJ_E, double *LJ_S, double *Box_x, double *Box_y, double *Box_z, double *cutoff, double *Pt_ePos_x, double *Pt_ePos_y, double *Pt_ePos_z, double *spr_k, double *Mass) {
	int i, LJ_pair;
	double Epair, Spair, Pairx, Pairy, Pairz, Dispair, Fpair, Atom_Fx, Atom_Fy, Atom_Fz;
	double Spring_Disx, Spring_Fx, Pt_Fx, Spring_Disy, Spring_Fy, Pt_Fy, Spring_Disz, Spring_Fz, Pt_Fz, Ar_Fx, Ar_Fy, Ar_Fz;
	int tid = threadIdx.x;

	if (tid<Pt_N + Ar_N) {
		Atom_Fx = 0.0;
		Atom_Fy = 0.0;
		Atom_Fz = 0.0;
		for (i = 0; i<Pt_N + Ar_N; i++) {
			if (tid<Pt_N && i<Pt_N) {
				LJ_pair = 1;
			}
			else if (tid >= Pt_N && i >= Pt_N) {
				LJ_pair = 0;
			}
			else {
				LJ_pair = 2;
			}
			Epair = LJ_E[LJ_pair];
			Spair = LJ_S[LJ_pair];
			//
			Pairx = All_Pos_x[tid] - All_Pos_x[i];
			Pairy = All_Pos_y[tid] - All_Pos_y[i];
			Pairz = All_Pos_z[tid] - All_Pos_z[i];
			if (abs(Pairx) >= Box_x[1] - Box_x[0] - (*cutoff)) {
				Pairx -= (Box_x[1] - Box_x[0])*Pairx / abs(Pairx);
			}
			if (abs(Pairy) >= Box_y[1] - Box_y[0] - (*cutoff)) {
				Pairy -= (Box_y[1] - Box_y[0])*Pairy / abs(Pairy);
			}
			//
			Dispair = sqrt(Pairx * Pairx + Pairy * Pairy + Pairz * Pairz);
			if (Dispair > 0 && Dispair <= (*cutoff)) {
				Fpair = 48 * Epair*(pow(Spair, 12) / pow(Dispair, 13) - 0.5*pow(Spair, 6) / pow(Dispair, 7));
				Atom_Fx += Pairx * Fpair / Dispair;
				Atom_Fy += Pairy * Fpair / Dispair;
				Atom_Fz += Pairz * Fpair / Dispair;
			}
		}
		if (tid<Pt_N) {
			//Pt
			Spring_Disx = All_Pos_x[tid] - Pt_ePos_x[tid];
			Spring_Fx = -(*spr_k) * Spring_Disx;
			Pt_Fx = Atom_Fx + Spring_Fx;
			All_Acc_x[tid] = Pt_Fx / Mass[1];
			Spring_Disy = All_Pos_y[tid] - Pt_ePos_y[tid];
			Spring_Fy = -(*spr_k) * Spring_Disy;
			Pt_Fy = Atom_Fy + Spring_Fy;
			All_Acc_y[tid] = Pt_Fy / Mass[1];
			Spring_Disz = All_Pos_z[tid] - Pt_ePos_z[tid];
			Spring_Fz = -(*spr_k) * Spring_Disz;
			Pt_Fz = Atom_Fz + Spring_Fz;
			All_Acc_z[tid] = Pt_Fz / Mass[1];
		}
		else {
			//Ar
			Ar_Fx = Atom_Fx;
			All_Acc_x[tid] = Ar_Fx / Mass[0];
			Ar_Fy = Atom_Fy;
			All_Acc_y[tid] = Ar_Fy / Mass[0];
			Ar_Fz = Atom_Fz;
			All_Acc_z[tid] = Ar_Fz / Mass[0];
		}
	}
}

/******************************************************************************/
__global__ void Verlet_Pos(double *All_Pos_x, double *All_Pos_y, double *All_Pos_z, double *All_Vel_x, double *All_Vel_y, double *All_Vel_z, double *All_Acc_x, double *All_Acc_y, double *All_Acc_z, double *dt) {
	int tid = threadIdx.x;

	if (tid<Pt_N + Ar_N) {
		All_Pos_x[tid] += All_Vel_x[tid] * (*dt) + 0.5*All_Acc_x[tid] * (*dt) * (*dt);
		All_Pos_y[tid] += All_Vel_y[tid] * (*dt) + 0.5*All_Acc_y[tid] * (*dt) * (*dt);
		All_Pos_z[tid] += All_Vel_z[tid] * (*dt) + 0.5*All_Acc_z[tid] * (*dt) * (*dt);
	}
}

/******************************************************************************/
__global__ void Verlet_Vel(double *All_Vel_x, double *All_Vel_y, double *All_Vel_z, double *All_Acc_temp_x, double *All_Acc_temp_y, double *All_Acc_temp_z, double *All_Acc_x, double *All_Acc_y, double *All_Acc_z, double *dt) {
	int tid = threadIdx.x;

	if (tid<Pt_N + Ar_N) {
		All_Vel_x[tid] += 0.5*(All_Acc_temp_x[tid] + All_Acc_x[tid])*(*dt);
		All_Vel_y[tid] += 0.5*(All_Acc_temp_y[tid] + All_Acc_y[tid])*(*dt);
		All_Vel_z[tid] += 0.5*(All_Acc_temp_z[tid] + All_Acc_z[tid])*(*dt);
	}
}

/******************************************************************************/
void Parameters::Dump(int All_type[], double All_Pos_x[], double All_Pos_y[], double All_Pos_z[], int timestep, int ds) {
	int i;

	if (timestep%ds == 0) {
		ofstream MD;
		MD.open("Kernel_MD_CUDA_C.dump", ios::app);
		MD << "ITEM: TIMESTEP\n";
		MD << timestep << "\n";
		MD << "ITEM: NUMBER OF ATOMS\n";
		MD << Pt_N + Ar_N << "\n";
		MD << "ITEM: BOX BOUNDS pp pp ff\n";
		MD << Box_x[0] << " " << Box_x[1] << "\n";
		MD << Box_y[0] << " " << Box_y[1] << "\n";
		MD << Box_z[0] << " " << Box_z[1] << "\n";
		MD << "ITEM: ATOMS id type x y z\n";
		for (i = 0; i < Pt_N + Ar_N; i++) {
			MD << i + 1 << " " << All_type[i] + 1 << " " << All_Pos_x[i] << " " << All_Pos_y[i] << " " << All_Pos_z[i] << "\n";
		}
		MD.close();
		ofstream Zt;
		Zt.open("Kernel_MD_CUDA_C_Zt.dat", ios::app);
		Zt << timestep * dt << " " << All_Pos_z[Pt_N] << "\n";
		Zt.close();
	}
}

/******************************************************************************/
void Parameters::Exit(int All_type[], double All_Pos_x[], double All_Pos_y[], double All_Pos_z[], int timestep) {

	if (All_Pos_z[Pt_N] > d || timestep >= Tt) {
		state = false;
		Dump(All_type, All_Pos_x, All_Pos_y, All_Pos_z, timestep);
	}
	else {
		Dump(All_type, All_Pos_x, All_Pos_y, All_Pos_z, timestep, dumpstep);
	}
}

/******************************************************************************/
double Parameters::random() {
	double R;
	R = 0.;
	while (R == 0.) {
		R = rand() / double(RAND_MAX);
	}
	return R;
}

////////////////////////////////////////////////////////////////////////////////
/*************************************main*************************************/
////////////////////////////////////////////////////////////////////////////////
int main() {
	class Parameters Pars;
	clock_t start, finish;
	double tperl;
	int All_type[Pt_N + Ar_N];
	double All_Pos_x[Pt_N + Ar_N], All_Pos_y[Pt_N + Ar_N], All_Pos_z[Pt_N + Ar_N], All_Vel_x[Pt_N + Ar_N], All_Vel_y[Pt_N + Ar_N], All_Vel_z[Pt_N + Ar_N], All_Acc_x[Pt_N + Ar_N], All_Acc_y[Pt_N + Ar_N], All_Acc_z[Pt_N + Ar_N];
	double *d_All_Pos_x, *d_All_Pos_y, *d_All_Pos_z, *d_All_Vel_x, *d_All_Vel_y, *d_All_Vel_z, *d_All_Acc_x, *d_All_Acc_y, *d_All_Acc_z, *d_All_Acc_temp_x, *d_All_Acc_temp_y, *d_All_Acc_temp_z, *d_Box_x, *d_Box_y, *d_Box_z, *d_Pt_ePos_x, *d_Pt_ePos_y, *d_Pt_ePos_z;
	double *d_dt, *d_Mass, *d_T, *d_LJ_E, *d_LJ_S, *d_cutoff, *d_spr_k, *d_Pt_argVel, *d_Pt_T;
	int timestep=0;

	Pars.Init();
	Pars.Initialization(All_type, All_Pos_x, All_Pos_y, All_Pos_z, All_Vel_x, All_Vel_y, All_Vel_z, All_Acc_x, All_Acc_y, All_Acc_z);
	Pars.Exit(All_type, All_Pos_x, All_Pos_y, All_Pos_z, timestep);
	//
	hipMalloc((void**)&d_All_Pos_x, sizeof(All_Pos_x));
	hipMalloc((void**)&d_All_Pos_y, sizeof(All_Pos_y));
	hipMalloc((void**)&d_All_Pos_z, sizeof(All_Pos_z));
	hipMalloc((void**)&d_All_Vel_x, sizeof(All_Vel_x));
	hipMalloc((void**)&d_All_Vel_y, sizeof(All_Vel_y));
	hipMalloc((void**)&d_All_Vel_z, sizeof(All_Vel_z));
	hipMalloc((void**)&d_All_Acc_x, sizeof(All_Acc_x));
	hipMalloc((void**)&d_All_Acc_y, sizeof(All_Acc_y));
	hipMalloc((void**)&d_All_Acc_z, sizeof(All_Acc_z));
	hipMalloc((void**)&d_All_Acc_temp_x, sizeof(All_Acc_x));
	hipMalloc((void**)&d_All_Acc_temp_y, sizeof(All_Acc_y));
	hipMalloc((void**)&d_All_Acc_temp_z, sizeof(All_Acc_z));
	hipMalloc((void**)&d_Box_x, sizeof(Pars.Box_x));
	hipMalloc((void**)&d_Box_y, sizeof(Pars.Box_y));
	hipMalloc((void**)&d_Box_z, sizeof(Pars.Box_z));
	hipMalloc((void**)&d_Pt_ePos_x, sizeof(Pars.Pt_ePos_x));
	hipMalloc((void**)&d_Pt_ePos_y, sizeof(Pars.Pt_ePos_y));
	hipMalloc((void**)&d_Pt_ePos_z, sizeof(Pars.Pt_ePos_z));
	hipMalloc((void**)&d_dt, sizeof(double));
	hipMalloc((void**)&d_Mass, sizeof(Pars.Mass));
	hipMalloc((void**)&d_T, sizeof(Pars.T));
	hipMalloc((void**)&d_LJ_E, sizeof(Pars.LJ_E));
	hipMalloc((void**)&d_LJ_S, sizeof(Pars.LJ_S));
	hipMalloc((void**)&d_cutoff, sizeof(double));
	hipMalloc((void**)&d_spr_k, sizeof(double));
	hipMalloc((void**)&d_Pt_argVel, sizeof(double));
	hipMalloc((void**)&d_Pt_T, sizeof(double));
	//
	hipMemcpy(d_All_Pos_x, All_Pos_x, sizeof(All_Pos_x), hipMemcpyHostToDevice);
	hipMemcpy(d_All_Pos_y, All_Pos_y, sizeof(All_Pos_y), hipMemcpyHostToDevice);
	hipMemcpy(d_All_Pos_z, All_Pos_z, sizeof(All_Pos_z), hipMemcpyHostToDevice);
	hipMemcpy(d_All_Vel_x, All_Vel_x, sizeof(All_Vel_x), hipMemcpyHostToDevice);
	hipMemcpy(d_All_Vel_y, All_Vel_y, sizeof(All_Vel_y), hipMemcpyHostToDevice);
	hipMemcpy(d_All_Vel_z, All_Vel_z, sizeof(All_Vel_z), hipMemcpyHostToDevice);
	hipMemcpy(d_All_Acc_x, All_Acc_x, sizeof(All_Acc_x), hipMemcpyHostToDevice);
	hipMemcpy(d_All_Acc_y, All_Acc_y, sizeof(All_Acc_y), hipMemcpyHostToDevice);
	hipMemcpy(d_All_Acc_z, All_Acc_z, sizeof(All_Acc_z), hipMemcpyHostToDevice);
	hipMemcpy(d_Box_x, Pars.Box_x, sizeof(Pars.Box_x), hipMemcpyHostToDevice);
	hipMemcpy(d_Box_y, Pars.Box_y, sizeof(Pars.Box_y), hipMemcpyHostToDevice);
	hipMemcpy(d_Box_z, Pars.Box_z, sizeof(Pars.Box_z), hipMemcpyHostToDevice);
	hipMemcpy(d_Pt_ePos_x, Pars.Pt_ePos_x, sizeof(Pars.Pt_ePos_x), hipMemcpyHostToDevice);
	hipMemcpy(d_Pt_ePos_y, Pars.Pt_ePos_y, sizeof(Pars.Pt_ePos_y), hipMemcpyHostToDevice);
	hipMemcpy(d_Pt_ePos_z, Pars.Pt_ePos_z, sizeof(Pars.Pt_ePos_z), hipMemcpyHostToDevice);
	hipMemcpy(d_dt, &Pars.dt, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_Mass, Pars.Mass, sizeof(Pars.Mass), hipMemcpyHostToDevice);
	hipMemcpy(d_T, Pars.T, sizeof(Pars.T), hipMemcpyHostToDevice);
	hipMemcpy(d_LJ_E, Pars.LJ_E, sizeof(Pars.LJ_E), hipMemcpyHostToDevice);
	hipMemcpy(d_LJ_S, Pars.LJ_S, sizeof(Pars.LJ_S), hipMemcpyHostToDevice);
	hipMemcpy(d_cutoff, &Pars.cutoff, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_spr_k, &Pars.spr_k, sizeof(double), hipMemcpyHostToDevice);
	start = clock();
	while (Pars.state) {
		//
		Verlet_Pos << <1, Pt_N + Ar_N >> >(d_All_Pos_x, d_All_Pos_y, d_All_Pos_z, d_All_Vel_x, d_All_Vel_y, d_All_Vel_z, d_All_Acc_x, d_All_Acc_y, d_All_Acc_z, d_dt);
		hipDeviceSynchronize();
		//
		Pos_period << <1, Pt_N + Ar_N >> >(d_All_Pos_x, d_All_Pos_y, d_Box_x, d_Box_y, d_Pt_ePos_x, d_Pt_ePos_y);
		hipDeviceSynchronize();
		//
		d_All_Acc_temp_x = d_All_Acc_x;
		d_All_Acc_temp_y = d_All_Acc_y;
		d_All_Acc_temp_z = d_All_Acc_z;
		//
		Acceleration_period << <1, Pt_N + Ar_N >> >(d_All_Pos_x, d_All_Pos_y, d_All_Pos_z, d_All_Acc_x, d_All_Acc_y, d_All_Acc_z, d_LJ_E, d_LJ_S, d_Box_x, d_Box_y, d_Box_z, d_cutoff, d_Pt_ePos_x, d_Pt_ePos_y, d_Pt_ePos_z, d_spr_k, d_Mass);
		hipDeviceSynchronize();
		//
		Verlet_Vel << <1, Pt_N + Ar_N >> >(d_All_Vel_x, d_All_Vel_y, d_All_Vel_z, d_All_Acc_temp_x, d_All_Acc_temp_y, d_All_Acc_temp_z, d_All_Acc_x, d_All_Acc_y, d_All_Acc_z, d_dt);
		hipDeviceSynchronize();
		//
		hipMemcpy(All_Vel_x, d_All_Vel_x, sizeof(All_Vel_x), hipMemcpyDeviceToHost);
		hipMemcpy(All_Vel_y, d_All_Vel_y, sizeof(All_Vel_y), hipMemcpyDeviceToHost);
		hipMemcpy(All_Vel_z, d_All_Vel_z, sizeof(All_Vel_z), hipMemcpyDeviceToHost);
		Pars.rescale_T1(All_Vel_x, All_Vel_y, All_Vel_z);
		hipMemcpy(d_Pt_argVel, Pars.Pt_argVel, sizeof(Pars.Pt_argVel), hipMemcpyHostToDevice);
		rescale_T2 << <1, Pt_N >> > (d_All_Vel_x, d_All_Vel_y, d_All_Vel_z, d_Pt_argVel);
		hipDeviceSynchronize();
		hipMemcpy(All_Vel_x, d_All_Vel_x, sizeof(All_Vel_x), hipMemcpyDeviceToHost);
		hipMemcpy(All_Vel_y, d_All_Vel_y, sizeof(All_Vel_y), hipMemcpyDeviceToHost);
		hipMemcpy(All_Vel_z, d_All_Vel_z, sizeof(All_Vel_z), hipMemcpyDeviceToHost);
		Pars.rescale_T3(All_Vel_x, All_Vel_y, All_Vel_z);
		hipMemcpy(d_Pt_T, &Pars.Pt_T, sizeof(double), hipMemcpyHostToDevice);
		rescale_T4 << <1, Pt_N >> > (d_All_Vel_x, d_All_Vel_y, d_All_Vel_z, d_T, d_Pt_T);
		hipDeviceSynchronize();
		//
		hipMemcpy(All_Pos_x, d_All_Pos_x, sizeof(All_Pos_x), hipMemcpyDeviceToHost);
		hipMemcpy(All_Pos_y, d_All_Pos_y, sizeof(All_Pos_y), hipMemcpyDeviceToHost);
		hipMemcpy(All_Pos_z, d_All_Pos_z, sizeof(All_Pos_z), hipMemcpyDeviceToHost);
		//
		Box_z[0]=All_Pos_z[0];
		for(i=0;i<Pt_N;i++){
			if(All_Pos_z[i]<Box_z[0]){
				Box_z[0]=All_Pos_z[i];
			}
		}
		//
		timestep += 1;
		Pars.Exit(All_type, All_Pos_x, All_Pos_y, All_Pos_z, timestep);
		finish = clock();
		tperl = double(finish - start) / CLOCKS_PER_SEC / timestep;
		cout << timestep << " TimeSteps; ArgTime: " << tperl << " Seconds!\r";
	}
	hipFree(d_All_Pos_x);
	hipFree(d_All_Pos_y);
	hipFree(d_All_Pos_z);
	hipFree(d_All_Vel_x);
	hipFree(d_All_Vel_y);
	hipFree(d_All_Vel_z);
	hipFree(d_All_Acc_x);
	hipFree(d_All_Acc_y);
	hipFree(d_All_Acc_z);
	hipFree(d_All_Acc_temp_x);
	hipFree(d_All_Acc_temp_y);
	hipFree(d_All_Acc_temp_z);
	hipFree(d_Box_x);
	hipFree(d_Box_y);
	hipFree(d_Box_z);
	hipFree(d_Pt_ePos_x);
	hipFree(d_Pt_ePos_y);
	hipFree(d_Pt_ePos_z);
	hipFree(d_dt);
	hipFree(d_Mass);
	hipFree(d_T);
	hipFree(d_LJ_E);
	hipFree(d_LJ_S);
	hipFree(d_cutoff);
	hipFree(d_spr_k);
	hipFree(d_Pt_argVel);
	hipFree(d_Pt_T);
	cout << "\n";
	system("pause");
	return 0;
}
